
#include <hip/hip_runtime.h>
#include <iostream>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

// #define N (8*8)
// #define THREADS_PER_BLOCK 8

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

void random_ints(int *a, int size) {
    for (int i = 0; i < size; i++) {
        a[i] = rand();
    }
}

int main(void) {
    srand(100);
    int *a, *b, *c; // host копии a, b, c
    int *dev_a, *dev_b, *dev_c; // device копии of a, b, c
    int size = N * sizeof(int);
    //выделяем память на device для of a, b, c
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    random_ints(a, N);
    random_ints(b, N);

    //копируем ввод на device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    //запускаем на выполнение add() kernel с блоками и тредами
    add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);
    // копируем результат работы device на host ( копия c )
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << "\n";

    free(a); free(b); free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[ threadIdx.x];
    // blockIdx.x blockIdx.x blockIdx.x
}

void random_ints(int *a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = rand();
    }
}


#define N 5

int main( void ) {
    srand(100);
    int *a, *b, *c; //host копии a, b, c
    int *dev_a, *dev_b, *dev_c; //device копии of a, b, c
    int size = N * sizeof( int );
    //выделяем память для копий a, b, c
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    random_ints(a, N);
    random_ints(b, N);
    // копируем ввод device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    // запускаем на выполнение add() kernel с N тредами в блоке

    add<<< 1, N >>>(dev_a, dev_b, dev_c);
    // N, 1
    // копируем результат работы device обратно на host (копия c)
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << "\n";
    free(a); free(b); free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
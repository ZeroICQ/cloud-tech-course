// Квадрат числа N


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 24
__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = a[i]*a[i];
    }
}

int main() {
    int ha[N], hb[N];
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));
    for (int i = 0; i<N; ++i) {
       ha[i] = i;
    }
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    add<<<N, 1>>>(da, db);
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }
    hipFree(da);
    hipFree(db);
    return 0;
}

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int *a, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = rand();
    }
}

#define N 5

int main( void ) {
    srand(100);
    int *a, *b, *c; // host копии a, b, c
    int *dev_a, *dev_b, *dev_c; // device копии a, b, c
    int size = N * sizeof(int);
    //выделяем память для device копий a, b, c
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);
    random_ints(a, N);
    random_ints(b, N);
    // копируем ввод на device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    // launch add() kernel with N parallel blocks
    add<<<N, 1>>>(dev_a, dev_b, dev_c);
    // копируем результат работы device обратно на host – копию c
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << "\n";
    free(a); free(b); free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}

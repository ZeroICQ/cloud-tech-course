
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void incKernel (float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = data[idx] + 1.0f;
}

int main (int argc, char * argv []) {
    int n = 16 * 1024 * 1024;
    int numBytes = n * sizeof (float);

    // выделение памяти на хосте
    float *a = new float[n];
    for (int i = 0; i < n; i++)
        a [i] = 0.0f;

    // выделение памяти на девайсе
    float *dev = NULL;
    hipMalloc((void**) &dev, numBytes);
    // Устоновка конфигурации запуска ядра
    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);
    // создание обработчиков событий cuda
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // асинхронно выдаем работу на GPU (все в поток 0)
    hipEventRecord(start, 0);
    hipMemcpy(dev, a, numBytes, hipMemcpyHostToDevice);
    incKernel<<<blocks, threads>>>(dev);
    hipMemcpy(a, dev, numBytes, hipMemcpyDeviceToHost);
    hipEventRecord (stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    // Печатаем время работы на CPU и GPU
    printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);
    // проверка аутпута на корректность
    printf("--------------------------------------------------------------\n");
    for (int i = 0; i < n; i++)
        if (a [i] != 1.0f) {
            printf ("Error in pos %d, %f\n", i, a[i]);
            break;
        }
    // освобождение ресурсов
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dev);
    delete a;
    return 0;
}
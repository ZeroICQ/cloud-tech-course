
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

__global__ void vecAdd_kernel(const float *a, const float *b, float *result, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        result[i] = a[i] + b[i];
}

int main() {
    int n = 100;
    float *a = new float[n], *a_gpu;

    hipMalloc((void**) &a_gpu, n * sizeof(float));
    float *b = new float[n], *b_gpu;
    hipMalloc((void**) &b_gpu, n * sizeof(float));

    float *result = new float[n], *result_gpu;
    hipMalloc((void**) &result_gpu, n * sizeof(float));

    for (int i = 0; i < n; i++)
        a[i] = b[i] = i;

    hipMemcpy(a_gpu, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, n * sizeof(float), hipMemcpyHostToDevice);

    const int block_size = 256;

    int num_blocks = (n + block_size - 1) / block_size;
    vecAdd_kernel <<<num_blocks, block_size>>>(a_gpu, b_gpu, result_gpu, n);

    hipMemcpy(result, result_gpu, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int x = 0; x < 10; x++)
        cout<<result[x]<<endl;

    delete[] a;
    delete[] b;
    delete[] result;
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(result_gpu);
    return 0;
}